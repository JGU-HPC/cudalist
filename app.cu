#include "hip/hip_runtime.h"
#include <iostream>
#include "cudalist.cuh"

#define GRIDDIM 256
#define BLOCKDIM 64

__global__
void internal_memory(float * result) {

    int thid = blockDim.x*blockIdx.x+threadIdx.x;
    float memory[BLOCKDIM];
    culist<float, int> list(memory, blockDim.x);
    
    for (int i = 0; i < 100000; ++i) {
    
        for (int m = 0; m < blockDim.x/2; ++m) {
            list.push_front(threadIdx.x+m);
            list.push_back(threadIdx.x+m);
        }
    
        for (int m = 0; m < blockDim.x/2; ++m) {
            list.pop_front();
            list.pop_back();
        }
    }
    
    for (int m = 0; m < blockDim.x/2; ++m) {
            list.push_front(threadIdx.x+m);
            list.push_back(threadIdx.x+m);
    }
    
    result[thid] = list[0];
}

__global__
void external_memory(float * result, float * memory) {

    int thid = blockDim.x*blockIdx.x+threadIdx.x;
    culist<float, int> list(memory+thid*blockDim.x, blockDim.x);
    
    for (int i = 0; i < 100000; ++i) {
    
        for (int m = 0; m < blockDim.x/2; ++m) {
            list.push_front(threadIdx.x+m);
            list.push_back(threadIdx.x+m);
        }
    
        for (int m = 0; m < blockDim.x/2; ++m) {
            list.pop_front();
            list.pop_back();
        }
    }
    
    for (int m = 0; m < blockDim.x/2; ++m) {
            list.push_front(threadIdx.x+m);
            list.push_back(threadIdx.x+m);
    }
    
    result[thid] = list[0];
}

__global__
void shared_memory(float * result) {

    int thid = blockDim.x*blockIdx.x+threadIdx.x;
    __shared__ float memory [BLOCKDIM*BLOCKDIM];
    culist<float, int> list(memory+threadIdx.x*blockDim.x, blockDim.x);
    
    for (int i = 0; i < 100000; ++i) {
    
        for (int m = 0; m < blockDim.x/2; ++m) {
            list.push_front(threadIdx.x+m);
            list.push_back(threadIdx.x+m);
        }
    
        for (int m = 0; m < blockDim.x/2; ++m) {
            list.pop_front();
            list.pop_back();
        }
    }
    
    for (int m = 0; m < blockDim.x/2; ++m) {
            list.push_front(threadIdx.x+m);
            list.push_back(threadIdx.x+m);
    }
    
    result[thid] = list[0];
}

int main() {

    float *Memory = NULL, *Result = NULL, *result = new float[GRIDDIM*BLOCKDIM];
    hipMalloc(&Memory, sizeof(float)*GRIDDIM*BLOCKDIM*BLOCKDIM);
    hipMalloc(&Result, sizeof(float)*GRIDDIM*BLOCKDIM);

    internal_memory<<<GRIDDIM, BLOCKDIM>>>(Result);                // fastest
    //external_memory<<<GRIDDIM, BLOCKDIM>>>(Result, Memory);      // meh
    //shared_memory<<<GRIDDIM, BLOCKDIM>>>(Result);                // meh^2

    hipMemcpy(result, Result, sizeof(float)*GRIDDIM*BLOCKDIM, 
               hipMemcpyDeviceToHost);
    
    for (int m = 0; m < GRIDDIM*BLOCKDIM; ++m)
        std::cout << result[m] << std::endl;
}
